#include "hip/hip_runtime.h"
#include "CudaBSOperator.h"
#include <thrust\device_vector.h>
#include <thrust\count.h>
#include "RandUtils.h"
#define BMSIZE 50
int width = 0;
int height = 0;
__constant__ uchar cpopcount_LUT8[256];
 texture<uchar4> ImageTexture;
#define TILE_W 16
#define TILE_H 16
#define R 2
#define BLOCK_W (TILE_W+(2*R))
#define BLOCK_H (TILE_H + (2*R))
//ȡ(x,y)���ص�id��ֵ��ÿ��������BMSIZE�����ݣ�����ͼ��������˳������, 
template<typename T>
 __device__ T& GetRefFromBigMatrix(PtrStep<T> mat, int width, int height, int id, int x, int y,int bmSize = 50)
{
	int col = ((y*width)+x)*bmSize+id;
	return mat(0,col);
}
 template<typename T>
 __device__ T* GetPointerFromBigMatrix(PtrStep<T> mat, int width, int height, int id, int x, int y,int bmSize = 50)
{
	int col = ((y*width)+x)*bmSize+id;
	return  (mat.data+col);
}

template<typename T>
__device__ T   GetValueFromBigMatrix(const PtrStep<T> mat, int width, int height, int id, int x, int y,int bmSize = 50)
{
	int col = ((y*width)+x)*bmSize+id;
	return mat(0,col);
}
template<typename T>
 __device__  void SetValueToBigMatrix(PtrStep<T> mat, int width, int height, int id, int x, int y,const T& v, int bmSize = 50)
{
	int col = ((y*width)+x)*bmSize+id;
	mat(0,col) = v;
}

__device__ size_t L1dist_uchar(const uchar4& a, const uchar4& b)
{
	return abs(a.x-b.x) + abs(a.y-b.y) + abs(a.z-b.z);
}
__device__ size_t absdiff_uchar(const uchar&a , const uchar& b)
{
	return abs((int)a - (int)b);
}
//! computes the population count of a 16bit vector using an 8bit popcount LUT (min=0, max=48)
__device__ uchar popcount_ushort_8bitsLUT(ushort x) {
	//! popcount LUT for 8bit vectors
	
	return cpopcount_LUT8[(uchar)x] + cpopcount_LUT8[(uchar)(x>>8)];
}
__device__ size_t hdist_ushort_8bitLUT(const ushort& a, const ushort& b)
{

	return popcount_ushort_8bitsLUT(a^b);
}
__device__ size_t hdist_ushort_8bitLUT(const ushort4& a, const ushort4& b)
{

	return popcount_ushort_8bitsLUT(a.x^b.x)+popcount_ushort_8bitsLUT(a.y^b.y)+popcount_ushort_8bitsLUT(a.z^b.z);
}


void InitConstantMem()
{
		const uchar hpopcount_LUT8[256] = {
		0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 4,
		1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
		1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
		2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
		1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
		2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
		2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
		3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
		1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
		2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
		2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
		3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
		2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
		3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
		3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
		4, 5, 5, 6, 5, 6, 6, 7, 5, 6, 6, 7, 6, 7, 7, 8,
	};
	hipMemcpyToSymbol(HIP_SYMBOL(cpopcount_LUT8),hpopcount_LUT8,sizeof(uchar)*256);
}

__device__ void LBSP(const PtrStep<uchar4>& img, const uchar4& color, const int x, const int y, const size_t* const t, ushort4& out)
{
	uchar4 p0 = img(1+y,x-1);
	uchar4 p1 = img(y-1,x+1);
	uchar4 p2 = img(y+1, x+1);
	uchar4 p3 = img(y-1,x-1);
	uchar4 p4 = img( y, x+1);
	uchar4 p5 = img( y-1,x);
	uchar4 p6 = img(y, x-1);
	uchar4 p7 = img( y+1, x);
	uchar4 p8 = img(y-2,x-2);
	uchar4 p9 = img( y+2, x+2);
	uchar4 p10 = img( y-2,x+2);
	uchar4 p11 = img(y+2,x- 2);
	uchar4 p12 = img( y+2, x);
	uchar4 p13 = img( y-2,x);
	uchar4 p14 = img( y, x+2);
	uchar4 p15 = img(y, x-2);
	out.x = ((absdiff_uchar(p0.x,color.x) > t[0]) << 15)
		+ ((absdiff_uchar(p1.x,color.x) > t[0]) << 14)
		+ ((absdiff_uchar(p2.x,color.x) > t[0]) << 13)
		+ ((absdiff_uchar(p3.x,color.x) > t[0]) << 12)
		+ ((absdiff_uchar(p4.x,color.x) > t[0]) << 11)
		+ ((absdiff_uchar(p5.x,color.x) > t[0]) << 10)
		+ ((absdiff_uchar(p6.x,color.x) > t[0]) << 9)
		+ ((absdiff_uchar(p7.x,color.x) > t[0]) << 8)
		+ ((absdiff_uchar(p8.x,color.x) > t[0]) << 7)
		+ ((absdiff_uchar(p9.x,color.x) > t[0]) << 6)
		+ ((absdiff_uchar(p10.x,color.x) > t[0]) << 5)
		+ ((absdiff_uchar(p11.x,color.x) > t[0]) << 4)
		+ ((absdiff_uchar(p12.x,color.x) > t[0]) << 3)
		+ ((absdiff_uchar(p13.x,color.x) > t[0]) << 2)
		+ ((absdiff_uchar(p14.x,color.x) > t[0]) << 1)
		+ ((absdiff_uchar(p15.x,color.x) > t[0]));
	out.y = ((absdiff_uchar(p0.y,color.y) > t[1]) << 15)
		+ ((absdiff_uchar(p1.y,color.y) > t[1]) << 14)
		+ ((absdiff_uchar(p2.y,color.y) > t[1]) << 13)
		+ ((absdiff_uchar(p3.y,color.y) > t[1]) << 12)
		+ ((absdiff_uchar(p4.y,color.y) > t[1]) << 11)
		+ ((absdiff_uchar(p5.y,color.y) > t[1]) << 10)
		+ ((absdiff_uchar(p6.y,color.y) > t[1]) << 9)
		+ ((absdiff_uchar(p7.y,color.y) > t[1]) << 8)
		+ ((absdiff_uchar(p8.y,color.y) > t[1]) << 7)
		+ ((absdiff_uchar(p9.y,color.y) > t[1]) << 6)
		+ ((absdiff_uchar(p10.y,color.y) > t[1]) << 5)
		+ ((absdiff_uchar(p11.y,color.y) > t[1]) << 4)
		+ ((absdiff_uchar(p12.y,color.y) > t[1]) << 3)
		+ ((absdiff_uchar(p13.y,color.y) > t[1]) << 2)
		+ ((absdiff_uchar(p14.y,color.y) > t[1]) << 1)
		+ ((absdiff_uchar(p15.y,color.y) > t[1]));
	out.z = ((absdiff_uchar(p0.z,color.z) > t[2]) << 15)
		+ ((absdiff_uchar(p1.z,color.z) > t[2]) << 14)
		+ ((absdiff_uchar(p2.z,color.z) > t[2]) << 13)
		+ ((absdiff_uchar(p3.z,color.z) > t[2]) << 12)
		+ ((absdiff_uchar(p4.z,color.z) > t[2]) << 11)
		+ ((absdiff_uchar(p5.z,color.z) > t[2]) << 10)
		+ ((absdiff_uchar(p6.z,color.z) > t[2]) << 9)
		+ ((absdiff_uchar(p7.z,color.z) > t[2]) << 8)
		+ ((absdiff_uchar(p8.z,color.z) > t[2]) << 7)
		+ ((absdiff_uchar(p9.z,color.z) > t[2]) << 6)
		+ ((absdiff_uchar(p10.z,color.z) > t[2]) << 5)
		+ ((absdiff_uchar(p11.z,color.z) > t[2]) << 4)
		+ ((absdiff_uchar(p12.z,color.z) > t[2]) << 3)
		+ ((absdiff_uchar(p13.z,color.z) > t[2]) << 2)
		+ ((absdiff_uchar(p14.z,color.z) > t[2]) << 1)
		+ ((absdiff_uchar(p15.z,color.z) > t[2]));
}
__device__ void LBSP(const uchar4* blockColor, const uchar4& color, const int x, const int y, int width,const size_t* const t, ushort4& out)
{
	int idx = (y+R)*width + x +R;
	uchar4 p0 = blockColor[idx+width-1];
	uchar4 p1 = blockColor[idx-width+1];
	uchar4 p2 = blockColor[idx+1+width];
	uchar4 p3 = blockColor[idx-1-width];
	uchar4 p4 = blockColor[ idx+1];
	uchar4 p5 = blockColor[idx-width];
	uchar4 p6 = blockColor[idx-1];
	uchar4 p7 = blockColor[idx+width];
	uchar4 p8 = blockColor[idx-2*width-2];
	uchar4 p9 = blockColor[idx+2*width+2];
	uchar4 p10 =blockColor[idx-2*width+2];
	uchar4 p11 = blockColor[idx+2*width-2];
	uchar4 p12 =blockColor[idx+2*width];
	uchar4 p13 =blockColor[idx-2*width];
	uchar4 p14 =blockColor[idx+2];
	uchar4 p15 = blockColor[idx-2];
	out.x = ((absdiff_uchar(p0.x,color.x) > t[0]) << 15)
		+ ((absdiff_uchar(p1.x,color.x) > t[0]) << 14)
		+ ((absdiff_uchar(p2.x,color.x) > t[0]) << 13)
		+ ((absdiff_uchar(p3.x,color.x) > t[0]) << 12)
		+ ((absdiff_uchar(p4.x,color.x) > t[0]) << 11)
		+ ((absdiff_uchar(p5.x,color.x) > t[0]) << 10)
		+ ((absdiff_uchar(p6.x,color.x) > t[0]) << 9)
		+ ((absdiff_uchar(p7.x,color.x) > t[0]) << 8)
		+ ((absdiff_uchar(p8.x,color.x) > t[0]) << 7)
		+ ((absdiff_uchar(p9.x,color.x) > t[0]) << 6)
		+ ((absdiff_uchar(p10.x,color.x) > t[0]) << 5)
		+ ((absdiff_uchar(p11.x,color.x) > t[0]) << 4)
		+ ((absdiff_uchar(p12.x,color.x) > t[0]) << 3)
		+ ((absdiff_uchar(p13.x,color.x) > t[0]) << 2)
		+ ((absdiff_uchar(p14.x,color.x) > t[0]) << 1)
		+ ((absdiff_uchar(p15.x,color.x) > t[0]));
	out.y = ((absdiff_uchar(p0.y,color.y) > t[1]) << 15)
		+ ((absdiff_uchar(p1.y,color.y) > t[1]) << 14)
		+ ((absdiff_uchar(p2.y,color.y) > t[1]) << 13)
		+ ((absdiff_uchar(p3.y,color.y) > t[1]) << 12)
		+ ((absdiff_uchar(p4.y,color.y) > t[1]) << 11)
		+ ((absdiff_uchar(p5.y,color.y) > t[1]) << 10)
		+ ((absdiff_uchar(p6.y,color.y) > t[1]) << 9)
		+ ((absdiff_uchar(p7.y,color.y) > t[1]) << 8)
		+ ((absdiff_uchar(p8.y,color.y) > t[1]) << 7)
		+ ((absdiff_uchar(p9.y,color.y) > t[1]) << 6)
		+ ((absdiff_uchar(p10.y,color.y) > t[1]) << 5)
		+ ((absdiff_uchar(p11.y,color.y) > t[1]) << 4)
		+ ((absdiff_uchar(p12.y,color.y) > t[1]) << 3)
		+ ((absdiff_uchar(p13.y,color.y) > t[1]) << 2)
		+ ((absdiff_uchar(p14.y,color.y) > t[1]) << 1)
		+ ((absdiff_uchar(p15.y,color.y) > t[1]));
	out.z = ((absdiff_uchar(p0.z,color.z) > t[2]) << 15)
		+ ((absdiff_uchar(p1.z,color.z) > t[2]) << 14)
		+ ((absdiff_uchar(p2.z,color.z) > t[2]) << 13)
		+ ((absdiff_uchar(p3.z,color.z) > t[2]) << 12)
		+ ((absdiff_uchar(p4.z,color.z) > t[2]) << 11)
		+ ((absdiff_uchar(p5.z,color.z) > t[2]) << 10)
		+ ((absdiff_uchar(p6.z,color.z) > t[2]) << 9)
		+ ((absdiff_uchar(p7.z,color.z) > t[2]) << 8)
		+ ((absdiff_uchar(p8.z,color.z) > t[2]) << 7)
		+ ((absdiff_uchar(p9.z,color.z) > t[2]) << 6)
		+ ((absdiff_uchar(p10.z,color.z) > t[2]) << 5)
		+ ((absdiff_uchar(p11.z,color.z) > t[2]) << 4)
		+ ((absdiff_uchar(p12.z,color.z) > t[2]) << 3)
		+ ((absdiff_uchar(p13.z,color.z) > t[2]) << 2)
		+ ((absdiff_uchar(p14.z,color.z) > t[2]) << 1)
		+ ((absdiff_uchar(p15.z,color.z) > t[2]));
}
__global__ void CudaBSOperatorKernel(const PtrStepSz<uchar4> img, double* homography, int frameIndex,
PtrStep<uchar4> colorModel,PtrStep<uchar4> wcolorModel,
PtrStep<ushort4> descModel,PtrStep<ushort4> wdescModel,
PtrStep<uchar> bModel,PtrStep<uchar> wbModel,
PtrStep<float> fModel,PtrStep<float> wfModel,
PtrStep<uchar> fgMask,	uchar* outMask, float fCurrLearningRateLowerCap,float fCurrLearningRateUpperCap, size_t* m_anLBSPThreshold_8bitLUT)
{
	
	__shared__ uchar4 scolor[BLOCK_W*BLOCK_H];
	int width = img.cols;
	int height = img.rows;
	// First batch loading
	int dest = threadIdx.y * TILE_W + threadIdx.x,
		destY = dest / BLOCK_W, destX = dest % BLOCK_W,
		srcY = blockIdx.y * TILE_W + destY - R,
		srcX = blockIdx.x * TILE_W + destX - R,
		src = (srcY * width + srcX);
	srcX = max(0,srcX);
	srcX = min(srcX,width-1);
	srcY = max(srcY,0);
	srcY = min(srcY,height-1);
	//scolor[dest] = img(srcY,srcX);
	scolor[dest] = tex1Dfetch(ImageTexture,srcY*width+srcX);

	//second batch loading
	dest = threadIdx.y * TILE_W + threadIdx.x + TILE_W * TILE_W;
	destY = dest / BLOCK_W, destX = dest % BLOCK_W;
	srcY = blockIdx.y * TILE_W + destY - R;
	srcX = blockIdx.x * TILE_W + destX - R;


	if (destY < BLOCK_W)
	{
		srcX = max(0,srcX);	 
		srcX = min(srcX,width-1);
		srcY = max(srcY,0);
		srcY = min(srcY,height-1);
		//scolor[destX + destY * BLOCK_W] = img(srcY,srcX);
		scolor[dest] = tex1Dfetch(ImageTexture,srcY*width+srcX);
	}
	__syncthreads();

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if(x < img.cols-2 && x>=2 && y>=2 && y < img.rows-2)
	{
		
		hiprandState state;
		hiprand_init(threadIdx.x,0,0,&state);
		double* ptr = homography;
		float fx,fy,fw;
		fx = x*ptr[0] + y*ptr[1] + ptr[2];
		fy = x*ptr[3] + y*ptr[4] + ptr[5];
		fw = x*ptr[6] + y*ptr[7] + ptr[8];
		fx /=fw;
		fy/=fw;
		int wx = (int)(fx+0.5);
		int wy = (int)(fy+0.5);
		
		float* fptr = GetPointerFromBigMatrix(fModel,width,height,0,x,y,10);
		float* pfCurrLearningRate = fptr;
		float* pfCurrDistThresholdFactor = fptr +1;
		float* pfCurrVariationFactor = fptr +2;
		float* pfCurrMeanLastDist =fptr + 3;
		float* pfCurrMeanMinDist_LT =fptr +4;
		float* pfCurrMeanMinDist_ST =fptr +5;
		float* pfCurrMeanRawSegmRes_LT = fptr + 6;
		float* pfCurrMeanRawSegmRes_ST = fptr + 7;
		float* pfCurrMeanFinalSegmRes_LT =fptr + 8;
		float* pfCurrMeanFinalSegmRes_ST =fptr + 9;
		uchar& pbUnstableRegionMask = GetRefFromBigMatrix(bModel,width,height,0,x,y,2);
		ushort4* anLastIntraDesc = descModel.data + 50*width*height + y*width + x;//desc model = 50 desc model + lastdesc
		uchar4* anLastColor =colorModel.data + 50*width*height + y*width + x;//desc model = 50 desc model + lastdesc//color model=50 bgmodel +  lastcolor
		
		fptr = GetPointerFromBigMatrix(wfModel,width,height,0,wx,wy,10);
		float* wpfCurrLearningRate = fptr;
		float* wpfCurrDistThresholdFactor = fptr +1;
		float* wpfCurrVariationFactor = fptr +2;
		float* wpfCurrMeanLastDist =fptr + 3;
		float* wpfCurrMeanMinDist_LT =fptr +4;
		float* wpfCurrMeanMinDist_ST =fptr +5;
		float* wpfCurrMeanRawSegmRes_LT = fptr + 6;
		float* wpfCurrMeanRawSegmRes_ST = fptr + 7;
		float* wpfCurrMeanFinalSegmRes_LT =fptr + 8;
		float* wpfCurrMeanFinalSegmRes_ST =fptr + 9;
		uchar& wpbUnstableRegionMask = GetRefFromBigMatrix(wbModel,width,height,0,x,y,2);
		ushort4* wanLastIntraDesc = wdescModel.data + 50*width*height + wy*width + wx;//desc model = 50 desc model + lastdesc
		uchar4* wanLastColor =wcolorModel.data + 50*width*height+ wy*width +wx;//desc model = 50 desc model + lastdesc//color model=50 bgmodel +  lastcolor
		

		

		ushort4* wBGIntraDescPtr = GetPointerFromBigMatrix(wdescModel,width,height,0,wx,wy);
		uchar4* wBGColorPtr = GetPointerFromBigMatrix(wcolorModel,width,height,0,wx,wy);
		ushort4*  BGIntraDescPtr = GetPointerFromBigMatrix(descModel,width,height,0,x,y);
		uchar4*  BGColorPtr= GetPointerFromBigMatrix(colorModel,width,height,0,x,y);
		

		
		unsigned idx = (threadIdx.y+R)*BLOCK_W + threadIdx.x+R;
		const uchar4 CurrColor = scolor[idx];
		//const uchar4 CurrColor = img(y,x);
		uchar anCurrColor[3] = {CurrColor.x,CurrColor.y,CurrColor.z};
		ushort4 CurrInterDesc, CurrIntraDesc;
		const size_t anCurrIntraLBSPThresholds[3] = {m_anLBSPThreshold_8bitLUT[CurrColor.x],m_anLBSPThreshold_8bitLUT[CurrColor.y],m_anLBSPThreshold_8bitLUT[CurrColor.z]};
		//LBSP(img,CurrColor,x,y,anCurrIntraLBSPThresholds,CurrIntraDesc);
		LBSP(scolor,CurrColor,threadIdx.x,threadIdx.y,BLOCK_W,anCurrIntraLBSPThresholds,CurrIntraDesc);

		outMask[y*width+x] = 0;
		//std::cout<<x<<","<<y<<std::endl;
		if (wx<2 || wx>= width-2 || wy<2 || wy>=height-2)
		{					
			//m_features.data[oidx_uchar] = 0xff;
			//m_nOutPixels ++;
			fgMask(y,x) = 0;
			outMask[y*width+x] = 0xff;
			size_t s_rand =hiprand(&state)%50;
			while(s_rand<50){
				BGIntraDescPtr[s_rand] = CurrIntraDesc;
				BGColorPtr[s_rand] = CurrColor;
				s_rand++;
			}
			return;
		}
		else
		{
			//���任
			ptr += 9;
			fx = x*ptr[0] + y*ptr[1] + ptr[2];
			fy = x*ptr[3] + y*ptr[4] + ptr[5];
			fw = x*ptr[6] + y*ptr[7] + ptr[8];
			fx /=fw;
			fy/=fw;
			//std::cout<<x<<","<<y<<std::endl;
			if (fx<2 || fx>= width-2 || fy<2 || fy>=height-2)
			{
				outMask[y*width+x] = 0xff;
			}
		}
		*pfCurrDistThresholdFactor =  *wpfCurrDistThresholdFactor;
		*pfCurrVariationFactor = *wpfCurrVariationFactor;
		*pfCurrLearningRate = *wpfCurrLearningRate;
		*pfCurrMeanLastDist = *wpfCurrMeanLastDist;
		*pfCurrMeanMinDist_LT = *wpfCurrMeanMinDist_LT;
		*pfCurrMeanMinDist_ST = *wpfCurrMeanMinDist_ST;
		*pfCurrMeanRawSegmRes_LT = *wpfCurrMeanRawSegmRes_LT; 
		*pfCurrMeanRawSegmRes_ST = *wpfCurrMeanRawSegmRes_ST;
		*pfCurrMeanFinalSegmRes_LT = *wpfCurrMeanFinalSegmRes_LT;
		*pfCurrMeanFinalSegmRes_ST = *wpfCurrMeanFinalSegmRes_ST;
		pbUnstableRegionMask = wpbUnstableRegionMask;


		for(int i=0; i<50; i++)
		{
			BGIntraDescPtr[i] = wBGIntraDescPtr[i];
			BGColorPtr[i] = wBGColorPtr[i];
		}
		/**anLastColor = *wanLastColor;
		*anLastIntraDesc = *wanLastIntraDesc;*/
		
		const float fRollAvgFactor_LT = 1.0f/min(frameIndex,25*4);
		const float fRollAvgFactor_ST = 1.0f/min(frameIndex,25);
		
		
		size_t nMinTotDescDist=48;
		size_t nMinTotSumDist=765;
		
		
		const size_t nCurrColorDistThreshold = (size_t)(((*wpfCurrDistThresholdFactor)*30)-((!wpbUnstableRegionMask)*6));
		size_t m_nDescDistThreshold = 3;
		const size_t nCurrDescDistThreshold = ((size_t)1<<((size_t)floor(*wpfCurrDistThresholdFactor+0.5f)))+m_nDescDistThreshold+(wpbUnstableRegionMask*m_nDescDistThreshold);
		const size_t nCurrTotColorDistThreshold = nCurrColorDistThreshold*3;
		const size_t nCurrTotDescDistThreshold = nCurrDescDistThreshold*3;
		const size_t nCurrSCColorDistThreshold = nCurrTotColorDistThreshold/2;


		
		
		ushort anCurrIntraDesc[3] = {CurrIntraDesc.x ,CurrIntraDesc.y, CurrIntraDesc.z};
		pbUnstableRegionMask = ((*wpfCurrDistThresholdFactor)>3.0 || (*wpfCurrMeanRawSegmRes_LT-*wpfCurrMeanFinalSegmRes_LT)>0.1 || (*wpfCurrMeanRawSegmRes_ST-*wpfCurrMeanFinalSegmRes_ST)>0.1)?1:0;
		size_t nGoodSamplesCount=0, nSampleIdx=0;

		
		while(nGoodSamplesCount<2 && nSampleIdx<50) {
			const ushort4 const BGIntraDesc = BGIntraDescPtr[nSampleIdx];
			const uchar4 const BGColor = BGColorPtr[nSampleIdx];
			
			uchar anBGColor[3] = {BGColor.x,BGColor.y,BGColor.z};
			ushort anBGIntraDesc[3] = {BGIntraDesc.x,BGIntraDesc.y,BGIntraDesc.z};
			const size_t anCurrInterLBSPThresholds[3] = {m_anLBSPThreshold_8bitLUT[BGColor.x],m_anLBSPThreshold_8bitLUT[BGColor.y],m_anLBSPThreshold_8bitLUT[BGColor.z]};
			//const size_t anCurrInterLBSPThresholds[3] = {m_anLBSPThreshold_8bitLUT[0],m_anLBSPThreshold_8bitLUT[0],m_anLBSPThreshold_8bitLUT[0]};
			
			LBSP(scolor,BGColor,threadIdx.x,threadIdx.y,BLOCK_W,anCurrInterLBSPThresholds,CurrInterDesc);
			ushort anCurrInterDesc[3] ={CurrInterDesc.x,CurrInterDesc.y, CurrInterDesc.z};
			
			size_t nTotDescDist = 0;
			size_t nTotSumDist = 0;
			for(size_t c=0;c<3; ++c) {
				const size_t nColorDist = abs(anCurrColor[c]-anBGColor[c]);
				
				if(nColorDist>nCurrSCColorDistThreshold)
					goto failedcheck3ch;
				size_t nInterDescDist = hdist_ushort_8bitLUT(anCurrInterDesc[c],anBGIntraDesc[c]);
				size_t nIntraDescDist = hdist_ushort_8bitLUT(anCurrIntraDesc[c],anBGIntraDesc[c]);
				const size_t nDescDist = (nIntraDescDist+nInterDescDist)/2;
				const size_t nSumDist = (nDescDist/2)*15+nColorDist;
				if(nSumDist>nCurrSCColorDistThreshold)
					goto failedcheck3ch;
				nTotDescDist += nDescDist;
				nTotSumDist += nSumDist;
				//nTotSumDist += nColorDist;
			}
			if(nTotDescDist>nCurrTotDescDistThreshold || nTotSumDist>nCurrTotColorDistThreshold)
				goto failedcheck3ch;

			if(nMinTotDescDist>nTotDescDist)
				nMinTotDescDist = nTotDescDist;
			if(nMinTotSumDist>nTotSumDist)
				nMinTotSumDist = nTotSumDist;
			nGoodSamplesCount++;
failedcheck3ch:
			nSampleIdx++;
		}
		//const float fNormalizedLastDist = ((float)L1dist_uchar(anLastColor,anCurrColor)/s_nColorMaxDataRange_3ch+(float)hdist_ushort_8bitLUT(anLastIntraDesc,anCurrIntraDesc)/s_nDescMaxDataRange_3ch)/2;
		const float fNormalizedLastDist = ((float)L1dist_uchar(*anLastColor,CurrColor)/765 +(float)hdist_ushort_8bitLUT(*anLastIntraDesc,CurrIntraDesc)/48)/2;		
		*pfCurrMeanLastDist = (*wpfCurrMeanLastDist)*(1.0f-fRollAvgFactor_ST) + fNormalizedLastDist*fRollAvgFactor_ST;
		if(nGoodSamplesCount<2) {
			// == foreground
			//const float fNormalizedMinDist = std::min(1.0f,((float)nMinTotSumDist/s_nColorMaxDataRange_3ch+(float)nMinTotDescDist/s_nDescMaxDataRange_3ch)/2 + (float)(m_nRequiredBGSamples-nGoodSamplesCount)/m_nRequiredBGSamples);
			const float fNormalizedMinDist = min(1.0f,((float)nMinTotSumDist/765) + (float)(2-nGoodSamplesCount)/2);
			*pfCurrMeanMinDist_LT = (*wpfCurrMeanMinDist_LT)*(1.0f-fRollAvgFactor_LT) + fNormalizedMinDist*fRollAvgFactor_LT;
			*pfCurrMeanMinDist_ST = (*wpfCurrMeanMinDist_ST)*(1.0f-fRollAvgFactor_ST) + fNormalizedMinDist*fRollAvgFactor_ST;
			*pfCurrMeanRawSegmRes_LT = (*wpfCurrMeanRawSegmRes_LT)*(1.0f-fRollAvgFactor_LT) + fRollAvgFactor_LT;
			*pfCurrMeanRawSegmRes_ST = (*wpfCurrMeanRawSegmRes_ST)*(1.0f-fRollAvgFactor_ST) + fRollAvgFactor_ST;
			fgMask(y,x) = UCHAR_MAX;
			if((hiprand(&state)%(size_t)2)==0) {
				const size_t s_rand = hiprand(&state)%50;
				BGIntraDescPtr[s_rand] = CurrIntraDesc;
				BGColorPtr[s_rand] = CurrColor;
			}
		}
		else {
			// == background
			fgMask(y,x) = 0;
			const float fNormalizedMinDist = ((float)nMinTotSumDist/765+(float)nMinTotDescDist/48)/2;
			//const float fNormalizedMinDist = ((float)nMinTotSumDist/765);
			*pfCurrMeanMinDist_LT = (*wpfCurrMeanMinDist_LT)*(1.0f-fRollAvgFactor_LT) + fNormalizedMinDist*fRollAvgFactor_LT;
			*pfCurrMeanMinDist_ST = (*wpfCurrMeanMinDist_ST)*(1.0f-fRollAvgFactor_ST) + fNormalizedMinDist*fRollAvgFactor_ST;
			*pfCurrMeanRawSegmRes_LT = (*wpfCurrMeanRawSegmRes_LT)*(1.0f-fRollAvgFactor_LT);
			*pfCurrMeanRawSegmRes_ST = (*wpfCurrMeanRawSegmRes_ST)*(1.0f-fRollAvgFactor_ST);
			const size_t nLearningRate =(size_t)ceil(*wpfCurrLearningRate);
			if(hiprand(&state)%nLearningRate==0) {
				const size_t s_rand =hiprand(&state)%50;
				BGIntraDescPtr[s_rand] = CurrIntraDesc;
				BGColorPtr[s_rand] = CurrColor;
			}
			int x_rand,y_rand;
			const bool bCurrUsing3x3Spread = !pbUnstableRegionMask;
			if(bCurrUsing3x3Spread)
			{
				getRandNeighborPosition_3x3(x_rand,y_rand,wx,wy,5/2,img.cols,img.rows);

				const size_t n_rand = hiprand(&state);
				const float fRandMeanLastDist = GetValueFromBigMatrix(wfModel,width,height,3,0,0);
				const float fRandMeanRawSegmRes = GetValueFromBigMatrix(wfModel,width,height,8,0,0);
				const size_t s_rand =hiprand(&state)%50;
				if((n_rand%(bCurrUsing3x3Spread?nLearningRate:(nLearningRate/2+1)))==0
					|| (fRandMeanRawSegmRes>0.995 && fRandMeanLastDist<0.01 && (n_rand%((size_t)fCurrLearningRateLowerCap))==0)) {
						SetValueToBigMatrix(colorModel,width,height,s_rand,x_rand,y_rand,CurrColor);
						SetValueToBigMatrix(descModel,width,height,s_rand,x_rand,y_rand,CurrIntraDesc);
				}
			}
		}
		float UNSTABLE_REG_RATIO_MIN = 0.1;
		float FEEDBACK_T_INCR = 0.5;
		float FEEDBACK_T_DECR = 0.1;
		float FEEDBACK_V_INCR(1.f);
		float FEEDBACK_V_DECR(0.1f);
		float FEEDBACK_R_VAR(0.01f);
		if(pbUnstableRegionMask || (min(*pfCurrMeanMinDist_LT,*pfCurrMeanMinDist_ST)<UNSTABLE_REG_RATIO_MIN && fgMask(y,x))) {
			if((*pfCurrLearningRate)<fCurrLearningRateUpperCap)
				*pfCurrLearningRate += FEEDBACK_T_INCR/(max(*pfCurrMeanMinDist_LT,*pfCurrMeanMinDist_ST)*(*pfCurrVariationFactor));
		}
		else if((*pfCurrLearningRate)>fCurrLearningRateLowerCap)
			*pfCurrLearningRate -= FEEDBACK_T_DECR*(*pfCurrVariationFactor)/max(*pfCurrMeanMinDist_LT,*pfCurrMeanMinDist_ST);
		if((*pfCurrLearningRate)< fCurrLearningRateLowerCap)
			*pfCurrLearningRate = fCurrLearningRateLowerCap;
		else if((*pfCurrLearningRate)>fCurrLearningRateUpperCap)
			*pfCurrLearningRate = fCurrLearningRateUpperCap;
		if(max(*pfCurrMeanMinDist_LT,*pfCurrMeanMinDist_ST)>UNSTABLE_REG_RATIO_MIN && GetValueFromBigMatrix(wbModel,width,height,1,wx,wy))
			(*pfCurrVariationFactor) += FEEDBACK_V_INCR;
		else if((*pfCurrVariationFactor)>FEEDBACK_V_DECR) {
			(*pfCurrVariationFactor) -= pbUnstableRegionMask?FEEDBACK_V_DECR/4:pbUnstableRegionMask?FEEDBACK_V_DECR/2:FEEDBACK_V_DECR;
			if((*pfCurrVariationFactor)<FEEDBACK_V_DECR)
				(*pfCurrVariationFactor) = FEEDBACK_V_DECR;
		}
		if((*pfCurrDistThresholdFactor)<pow(1.0f+min(*pfCurrMeanMinDist_LT,*pfCurrMeanMinDist_ST)*2,2))
			(*pfCurrDistThresholdFactor) += FEEDBACK_R_VAR*(*pfCurrVariationFactor-FEEDBACK_V_DECR);
		else {
			(*pfCurrDistThresholdFactor) -= FEEDBACK_R_VAR/(*pfCurrVariationFactor);
			if((*pfCurrDistThresholdFactor)<1.0f)
				(*pfCurrDistThresholdFactor) = 1.0f;
		}
		/*if(popcount_ushort_8bitsLUT(anCurrIntraDesc)>=4)
		++nNonZeroDescCount;*/
		*anLastColor = CurrColor;
		*anLastIntraDesc = CurrIntraDesc;

	}
	else if(x<width && y<height)
	{
		fgMask(y,x) = 0;
		outMask[y*width+x] =0;
	}
}

__global__ void CudaRefreshModelKernel(float refreshRate, int width ,int height,PtrStep<uchar> mask, PtrStep<uchar4> colorModels,PtrStep<ushort4> descModels, int modelSize,
	cv::gpu::PtrStep<float> fModel, cv::gpu::PtrStep<uchar> bModel)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width-2 && x>=2 && y>=2 && y <height-2 && mask(y,x) == 0xff)
	{
		hiprandState state;
		hiprand_init(threadIdx.x,0,0,&state);
		const size_t nBGSamplesToRefresh = refreshRate<1.0f?(size_t)(refreshRate*modelSize):modelSize;
		const size_t nRefreshStartPos = refreshRate<1.0f?hiprand(&state)%modelSize:0;
		size_t offset = width*height*modelSize;
		uchar4* colorPtr = colorModels.data + offset;
		ushort4* descPtr= descModels.data + offset;
		for(size_t s=nRefreshStartPos; s<nRefreshStartPos+nBGSamplesToRefresh; ++s) {

			int y_sample, x_sample;
			getRandSamplePosition(s,x_sample,y_sample,x,y,2,width,height);
			int idx =  y_sample*width+ x_sample;
			uchar4 value =  colorPtr[idx];
			ushort4 svalue = descPtr[idx];
			int pos = s%modelSize;
			SetValueToBigMatrix(colorModels,width,height,pos,x,y,value);
			SetValueToBigMatrix(descModels,width,height,pos,x,y,svalue);
		}
		const int fSize =10;
		const int bSize = 2;
		float* fptr = GetPointerFromBigMatrix(fModel,width,height,0,x,y,fSize);
		uchar* bptr = GetPointerFromBigMatrix(bModel,width,height,0,x,y,bSize);
		fptr[0] = 2.f;
		fptr[1] = 1.f;
		fptr[2] = 10.f;
		for( int i=3; i<fSize; i++)
		{
			fptr[i] = 0;
		}
		for(int i=0; i<bSize; i++)
		{
			bptr[i] = 0;
		}
	}

}

__global__ void CudaRefreshModelKernel(float refreshRate, int width ,int height,PtrStep<uchar4> colorModels,PtrStep<ushort4> descModels, int modelSize,
	cv::gpu::PtrStep<float> fModel, cv::gpu::PtrStep<uchar> bModel)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width-2 && x>=2 && y>=2 && y <height-2)
	{
		hiprandState state;
		hiprand_init(threadIdx.x,0,0,&state);
		const size_t nBGSamplesToRefresh = refreshRate<1.0f?(size_t)(refreshRate*modelSize):modelSize;
		const size_t nRefreshStartPos = refreshRate<1.0f?hiprand(&state)%modelSize:0;
		size_t offset = width*height*modelSize;
		uchar4* colorPtr = colorModels.data + offset;
		ushort4* descPtr= descModels.data + offset;
		for(size_t s=nRefreshStartPos; s<nRefreshStartPos+nBGSamplesToRefresh; ++s) {

			int y_sample, x_sample;
			getRandSamplePosition(s,x_sample,y_sample,x,y,2,width,height);
			int idx =  y_sample*width+ x_sample;
			uchar4 value =  colorPtr[idx];
			ushort4 svalue = descPtr[idx];
			int pos = s%modelSize;
			SetValueToBigMatrix(colorModels,width,height,pos,x,y,value);
			SetValueToBigMatrix(descModels,width,height,pos,x,y,svalue);
		}
		const int fSize =10;
		const int bSize = 2;
		float* fptr = GetPointerFromBigMatrix(fModel,width,height,0,x,y,fSize);
		uchar* bptr = GetPointerFromBigMatrix(bModel,width,height,0,x,y,bSize);
		fptr[0] = 2.f;
		fptr[1] = 1.f;
		fptr[2] = 10.f;
		for( int i=3; i<fSize; i++)
		{
			fptr[i] = 0;
		}
		for(int i=0; i<bSize; i++)
		{
			bptr[i] = 0;
		}
	}

}

__global__ void SCudaRefreshModelKernel(float refreshRate,const PtrStepSz<uchar4> lastImg,const PtrStepSz<ushort4> lastDescImg,PtrStep<uchar4>* colorModels,PtrStep<ushort4>* descModels, int modelSize)
{
	__shared__ uchar4 scolor[BLOCK_W*BLOCK_H];
	__shared__ ushort4 sdesc[BLOCK_W*BLOCK_H];
	int width = lastImg.cols;
	int height = lastImg.rows;
	// First batch loading
	int dest = threadIdx.y * TILE_W + threadIdx.x,
		destY = dest / BLOCK_W, destX = dest % BLOCK_W,
		srcY = blockIdx.y * TILE_W + destY - R,
		srcX = blockIdx.x * TILE_W + destX - R,
		src = (srcY * width + srcX);
	srcX = max(0,srcX);
	srcX = min(srcX,width-1);
	srcY = max(srcY,0);
	srcY = min(srcY,height-1);
	scolor[dest] = lastImg(srcY,srcX);
	sdesc[dest] = lastDescImg(srcY,srcX);
	//second batch loading
	dest = threadIdx.y * TILE_W + threadIdx.x + TILE_W * TILE_W;
	destY = dest / BLOCK_W, destX = dest % BLOCK_W;
	srcY = blockIdx.y * TILE_W + destY - R;
	srcX = blockIdx.x * TILE_W + destX - R;


	if (destY < BLOCK_W)
	{
		srcX = max(0,srcX);	 
		srcX = min(srcX,width-1);
		srcY = max(srcY,0);
		srcY = min(srcY,height-1);
		scolor[destX + destY * BLOCK_W] = lastImg(srcY,srcX);
		sdesc[destX + destY * BLOCK_W] = lastDescImg(srcY,srcX);
	}

	__syncthreads();

	int y = blockIdx.y * TILE_W + threadIdx.y;
	int  x = blockIdx.x * TILE_W + threadIdx.x;
	if(x < lastImg.cols && y < lastImg.rows)
	{

		hiprandState state;
		hiprand_init(threadIdx.x,0,0,&state);

		const size_t nBGSamplesToRefresh = refreshRate<1.0f?(size_t)(refreshRate*modelSize):modelSize;
		const size_t nRefreshStartPos = refreshRate<1.0f?hiprand(&state)%modelSize:0;
		for(size_t s=nRefreshStartPos; s<nRefreshStartPos+nBGSamplesToRefresh; ++s) {

			int y_sample, x_sample;
			getRandSamplePosition(s,x_sample,y_sample,x,y,2,width,height);
			y_sample -= y;
			x_sample -= x;
			//unsigned idx = bindex + x_sample + (y_sample*blockDim.x);
			unsigned idx = (threadIdx.y+R+y_sample)*BLOCK_W + threadIdx.x+R+x_sample;
			colorModels[s%modelSize](y,x) = scolor[idx];
			descModels[s%modelSize](y,x) = sdesc[idx];
		}
	}
	//__syncthreads();
}


__global__ void DownloadKernel(int width, int height, int id, const PtrStep<ushort4> models, int modelSize, PtrStep<ushort4> model)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width-2 && x>=2 && y>=2 && y <height-2)
	{
		
		model(y,x) = GetValueFromBigMatrix(models,width,height,id,x,y,modelSize);
		
	}
}
__global__ void DownloadColorKernel(int width, int height, int id, const PtrStep<uchar4> models, int modelSize, PtrStep<uchar4> model)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width-2 && x>=2 && y>=2 && y <height-2)
	{
		model(y,x) = GetValueFromBigMatrix(models,width,height,id,x,y,modelSize);
	}
}
void DownloadColorModel(int width,int height, cv::gpu::GpuMat& models, int size, int id, cv::gpu::GpuMat& model)
{
	dim3 block(16,16);
	dim3 grid((width + block.x - 1)/block.x,(height + block.y - 1)/block.y);
	DownloadColorKernel<<<grid,block>>>(width,height,id,models,size,model);
}
void DownloadModel(int width,int height,cv::gpu::GpuMat& models, int size, int id, cv::gpu::GpuMat& model)
{
	dim3 block(16,16);
	dim3 grid((width + block.x - 1)/block.x,(height + block.y - 1)/block.y);
	DownloadKernel<<<grid,block>>>(width,height,id,models,size,model);
}
void CudaBSOperator(const cv::gpu::GpuMat& img,double* homography, int frameIdx, 
PtrStep<uchar4> colorModel,PtrStep<uchar4> wcolorModel,
PtrStep<ushort4> descModel,PtrStep<ushort4> wdescModel,
PtrStep<uchar> bModel,PtrStep<uchar> wbModel,
PtrStep<float> fModel,PtrStep<float> wfModel,
PtrStep<uchar> fgMask,	uchar* outMask, float fCurrLearningRateLowerCap,float fCurrLearningRateUpperCap, size_t* m_anLBSPThreshold_8bitLUT)
{
	dim3 block(16,16);
	dim3 grid((img.cols + block.x - 1)/block.x,(img.rows + block.y - 1)/block.y);
	hipBindTexture( NULL, ImageTexture,
		img.ptr<uchar4>(),	sizeof(uchar4)*img.cols*img.rows );
	CudaBSOperatorKernel<<<grid,block>>>(img,homography,frameIdx,colorModel,
		wcolorModel,descModel, wdescModel,
		bModel,wbModel,fModel,wfModel,
		fgMask, outMask,fCurrLearningRateLowerCap, fCurrLearningRateUpperCap,  m_anLBSPThreshold_8bitLUT);
}
void CudaRefreshModel(float refreshRate,int width, int height,cv::gpu::GpuMat& mask, cv::gpu::GpuMat& colorModels, cv::gpu::GpuMat& descModels, 
	GpuMat& fModel, GpuMat& bModel)
{
	dim3 block(16,16);
	dim3 grid((width + block.x - 1)/block.x,(height + block.y - 1)/block.y);
	//colorModels������ downsample ��lastcolor
	//CudaRefreshModelKernel<<<grid,block>>>(refreshRate,lastImg,lastDescImg,ptr_colorModel,ptr_descModel,d_colorModels.size()-2);
	//colorModels������ downsample ��lastcolor
	CudaRefreshModelKernel<<<grid,block>>>(refreshRate,width,height,mask,colorModels,descModels,50,fModel,bModel);
}
void CudaRefreshModel(float refreshRate,int width, int height, cv::gpu::GpuMat& colorModels, cv::gpu::GpuMat& descModels, 
	GpuMat& fModel, GpuMat& bModel)
{
	dim3 block(16,16);
	dim3 grid((width + block.x - 1)/block.x,(height + block.y - 1)/block.y);
	//colorModels������ downsample ��lastcolor
	//CudaRefreshModelKernel<<<grid,block>>>(refreshRate,lastImg,lastDescImg,ptr_colorModel,ptr_descModel,d_colorModels.size()-2);
	//colorModels������ downsample ��lastcolor
	CudaRefreshModelKernel<<<grid,block>>>(refreshRate,width,height,colorModels,descModels,50,fModel,bModel);


}
bool equalToFF(uchar a)
{
	return a==0xff;
}
int CountOutPixel(const uchar* outMask,size_t size)
{

	
	return thrust::count(outMask,outMask+size,0);
}
//__global__ void testRandomKernel(int n, int* d_in, int* d_out)
//{
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//	if (idx >= n/2)
//		return;
//	int x = d_in[idx*2];
//	int y = d_in[idx*2+1];
//	int x_sample,y_sample;
//	getRandSamplePosition(x_sample,y_sample,x,y,2,100,100);
//	d_out[idx*2] =x_sample;
//	d_out[idx*2+1] =y_sample;
//
//}
//
//void testRandom()
//{
//	const int n = 100;
//	int *d_in,*d_out;
//	hipMalloc(&d_in,sizeof(int)*n);
//	hipMalloc(&d_out, sizeof(int)*n);
//	
//	int h_in[n],h_out[n];
//	for(int i=0; i<n; i+=2)
//	{
//		h_in[i] = i;
//		h_in[i+1]=i+1;
//	}
//	hipMemcpy(d_in,h_in,sizeof(int)*n,hipMemcpyHostToDevice);
//	testRandomKernel<<<n/2+127/128,128>>>(n,d_in,d_out);
//	hipMemcpy(h_out,d_out,sizeof(int)*n,hipMemcpyDeviceToHost);
//	for(int i=0; i<n; i+=2)
//		std::cout<<h_out[i]<<","<<h_out[i+1]<<std::endl;
//
//	for(int i=0; i<n; i+=2)
//	{
//		getRandSamplePosition(h_out[i],h_out[i+1],h_in[i],h_in[i+1],2,cv::Size(100,100));
//	}
//	std::cout<<"----------------------\n";
//	for(int i=0; i<n; i+=2)
//		std::cout<<h_out[i]<<","<<h_out[i+1]<<std::endl;
//}