#include "hip/hip_runtime.h"
#include "CudaSuperpixel.h"

__global__ void kInitClusterCentersKernel( float4* floatBuffer, int nWidth, int nHeight, int nSegs, SLICClusterCenter* vSLICCenterList )
{


	int blockWidth=nWidth/blockDim.x;
	int blockHeight=nHeight/gridDim.x;

	int clusterIdx=blockIdx.x*blockDim.x+threadIdx.x;
	int offsetBlock = blockIdx.x * blockHeight * nWidth + threadIdx.x * blockWidth;

	float2 avXY;

	avXY.x=threadIdx.x*blockWidth + (float)blockWidth/2.0;
	avXY.y=blockIdx.x*blockHeight + (float)blockHeight/2.0;

	//use a single point to init center
	int offset=offsetBlock + blockHeight/2 * nWidth+ blockWidth/2 ;

	float4 fPixel=floatBuffer[offset];
	float4 tmp;
	tmp.x = 0;
	tmp.y =0; 
	tmp.z = 0;

	for(int i=0; i<blockHeight*blockWidth; i++)
	{	
		tmp.x = tmp.x + floatBuffer[offset+i].x;
		tmp.y = tmp.y + floatBuffer[offset+i].y;
		tmp.z = tmp.z + floatBuffer[offset+i].z;
	}

	double sz = blockWidth * blockHeight;
	tmp.x = tmp.x / sz;
	tmp.y = tmp.y /sz;
	tmp.z = tmp.z/sz;
	

	vSLICCenterList[clusterIdx].rgb= tmp;
	vSLICCenterList[clusterIdx].xy=avXY;
	vSLICCenterList[clusterIdx].nPoints=0;
	
}


void InitClusterCenters(float4* d_rgbaBuffer, int width, int height, int step, int &nSeg, SLICClusterCenter* d_centers)
{
	dim3 blockDim = (width+ step-1) / step ;
	dim3 gridDim = (height + step -1) / step;
	nSeg = blockDim.x * gridDim.x;
	
	kInitClusterCentersKernel<<<gridDim,blockDim>>>(d_rgbaBuffer,width,height,nSeg,d_centers);
}